
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(const int *a, const int *b, int *c) {
  *c += *a + *b;
}

int main() {
  int a, b, c;
  int *a_ptr, *b_ptr, *c_ptr;
  const int size = sizeof(int);

  hipMalloc((void **) &a_ptr, size);
  hipMalloc((void **) &b_ptr, size);
  hipMalloc((void **) &c_ptr, size);

  a = 2;
  b = 3;
  c = 5;

  hipMemcpy(a_ptr, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(b_ptr, &b, size, hipMemcpyHostToDevice);
  hipMemcpy(c_ptr, &c, size, hipMemcpyHostToDevice);

  add<<<1,1>>>(a_ptr, b_ptr, c_ptr);

  hipMemcpy(&c, c_ptr, size, hipMemcpyDeviceToHost);

  printf("The sum is: %d\n", c);
  return 0;
}
